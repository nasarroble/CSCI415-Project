
#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
#include<cstring>
#include<ctime>
#include<cstdlib>
#include<sys/time.h>
#include<stdio.h>
#include<iomanip>
/* we need these includes for CUDA's random number stuff */
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
using namespace std;


#define MAX 26

 //array of all possible password characters
int b[1000]; //array of attempted password cracks
unsigned long long tries = 0;
char alphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };

size_t result = 1000 * sizeof(float);

int *a = (int *) malloc(result);

void serial_passwordCrack(int length){
bool cracked = false;
do{
    b[0]++;
    for(int i =0; i<length; i++){
        if (b[i] >= 26 + alphabet[i]){ 
            b[i] -= 26; 
            b[i+1]++;
        }else break;
    }
    cracked=true;
    for(int k=0; k<length; k++)
        if(b[k]!=a[k]){
            cracked=false;
            break;
        }
    if( (tries & 0x7ffffff) == 0 )
        cout << "\r       \r   ";
    else if( (tries & 0x1ffffff) == 0 )
        cout << ".";
    tries++;
}while(cracked==false);

}


__global__ void parallel_passwordCrack(int length,int*d_output,int *a, long attempts )
{	
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	bool cracked = false;
	int mark=0;
        char alphabetTable[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };        
	int newB[1000]; 

 // // randoms(states,alphabetTable,d_output);

 //  char alph ;//= 'a';
// while(!cracked){
//       alph =alphabetTable[rand()%26];
//        d_output[idx] = int(alph);
//      __syncthreads();
//       for(int i = 0; i< length; i++){
//         if(d_output[i] != a[i])
//         {
//           cracked = false;

//         }
//         else{
//           cracked = true;
//         }
//       }
//   }

__shared__ int nIter;
__shared__ int idT;


do{


//     newB[0]++;
        
//     if(mark<length){
//         if (newB[idx] >= 26 + alphabetTable[idx]){ 
//             newB[idx] -= 26; 
//             newB[idx+1]++;
//     }
// }else{
//         mark++;
//     }

   newB[0]++;
    for(int i =0; i<length; i++){
        if (newB[i] >= 26 + alphabetTable[i]){ 
            newB[i] -= 26; 
            newB[i+1]++;
        }else break;
    }
    
    cracked=true;
  //  nIter = 1;
    for(int k=0; k<length; k++)
    {
        if(newB[k]!=a[k]){
            cracked=false;
         //   nIter = 0;
            break;
        }else
        {
            cracked = true;
           // nIter = 1;
            // printf("idx:  %d  found\n", idx);
            //  d_output[k] = newB[k];

        }
    }
    if(cracked){
      __syncthreads();
      idT = idx;
      nIter = 1;
       __syncthreads();
      break;

    }
//    if( (tries & 0x7ffffff) == 0 )
//        cout << "\r       \r   ";
//    else if( (tries & 0x1ffffff) == 0 )
//        cout << ".";
    attempts++;
}while(!cracked);

if(idx == idT){
        for(int i = 0; i< length; i++){
  
             d_output[i] = newB[i];
    }


}

//newB[idx];


//    if( idx == 2 ){
//          nIter =idx+1;
//          printf("idx: %d: found, %d\n", idx, nIter);
//        }
// if(!nIter){
//   printf("idx: %d: not found \n", idx);
// }


}


long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}



int main()
{
int length; //length of password
int random; //random password to be generated
long attempts = 0; //number of attempts to crack the password
int *d_input = (int *) malloc(result);;


cout << "Enter a password length: ";
cin >> length;
int *h_gpu_result = (int*)malloc(1000*sizeof(int));

srand(time(NULL));
cout << "Random generated password: " << endl;
for (int i =0; i<length; i++){
    
        random = alphabet[(rand()%26)]; 
    
    a[i] = random; //adding random password to array
  //  d_input[i] = a[i];
    cout << char(a[i]);
}cout << "\n" << endl;

//declare GPU memory pointers
  int *d_output;
//allocate GPU memory
  hipMalloc((void **) &d_output,1000*sizeof(int));
  hipMalloc((void **) &d_input,result);
//transfer the array to the GP

  hipError_t err = hipSuccess;
err = hipMemcpy(d_input, a,result,hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy d_S from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
//launch the kernel
int threards = length*10;//(length*1000)/1024;


   // for(int i = 0; i< length; i++){
   //  printf("value: %d\n", d_input[i]);
   // }
//    

/* CUDA's random number library uses curandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t* states;

  /* allocate space on the GPU for the random states */
//  cudaMalloc((void**) &states, threards * sizeof(curandState_t));
    /* invoke the GPU to initialize all of the random states */
//   init<<<threards, threards>>>(time(0), states);


//parallel_passwordCrack<<<threards,1024>>>(length,d_output,d_input,attempts);
parallel_passwordCrack<<<1,threards>>>(length,d_output,d_input,attempts);
//copy back the result array to the CPU
hipMemcpy(h_gpu_result,d_output,1000*sizeof(int),hipMemcpyDeviceToHost);

// cout << "Serial Password Cracked: " << endl;
// serial_passwordCrack(length);
// cout << "\n";
// for(int i=0; i<length; i++){
//     cout << char(b[i]);
// }cout << "\nNumber of tries: " << tries << endl;

cout << "\nParallel Password Cracked: " << endl;
for(int i=0; i<length; i++){
//	cout << char(h_gpu_result[i]);
    printf("%c\n", char(h_gpu_result[i]));
}
cout << "\nNumber of attempts: " << attempts << endl;

hipFree(d_output);
hipFree(d_input);
hipFree(states);
free(h_gpu_result);

return 0;
}
